#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "reference_calc.cpp"
#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  int ridx = blockIdx.x * blockDim.x + threadIdx.x;
  int cidx = blockIdx.y * blockDim.y + threadIdx.y;
  if (ridx >= numRows || cidx >= numCols)
    return;
  int idx = ridx * numCols + cidx;
  uchar4 pixel = rgbaImage[idx];
  greyImage[idx] = .299f * pixel.x + .587f * pixel.y + .114f * pixel.z;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(32, 32, 1);  // Limit of #threads = 1024
  const dim3 gridSize((numRows + 31) / 32, (numCols + 31) / 32, 1);  // ceil(# / 32)
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  /****************************************************************************
  * You can use the code below to help with debugging, but make sure to       *
  * comment it out again before submitting your assignment for grading,       *
  * otherwise this code will take too much time and make it seem like your    *
  * GPU implementation isn't fast enough.                                     *
  *                                                                           *
  * This code generates a reference image on the host by running the          *
  * reference calculation we have given you.  It then copies your GPU         *
  * generated image back to the host and calls a function that compares the   *
  * the two and will output the first location they differ by too much.       *
  * ************************************************************************* */


  /*unsigned char *h_greyImageGPU = new unsigned char[numRows * numCols];
  unsigned char *h_greyImageRef = new unsigned char[numRows * numCols];

  checkCudaErrors(hipMemcpy(h_greyImageGPU, d_greyImage, 
                             numRows * numCols * sizeof(unsigned char), 
                             hipMemcpyDeviceToHost));
  referenceCalculation(h_rgbaImage, h_greyImageRef, numRows, numCols);

  checkResultsEps(h_greyImageRef, h_greyImageGPU, numRows * numCols, 1, .001); 
 
  delete [] h_greyImageGPU;
  delete [] h_greyImageRef;*/
}
