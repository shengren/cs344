#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"

#include <cstdio>

__global__ void ReduceOnce(float* const d,
                           const size_t n,
                           const size_t w,
                           const int op) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx >= w)
    return;

  float a = d[tidx];
  if (tidx + w < n) {
    float b = d[tidx + w];
    a = (op == 0) ? min(a, b) : max(a, b);
  }
  d[tidx] = a;
}

__global__ void BuildHistogram(const float* const d_val,
                               const size_t num_val,
                               unsigned int* const d_bin,
                               const size_t num_bin,
                               const float min_val,
                               const float range) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tidx >= num_val)
    return;

  unsigned int bin_idx =
    min((unsigned int)(num_bin - 1),
        (unsigned int)((d_val[tidx] - min_val) / range * num_bin));
  atomicAdd(&d_bin[bin_idx], 1);
}

__global__ void ExclusivePrefixSum(const unsigned int* const d_histo,
                                   unsigned int* const d_cdf,
                                   const size_t num_bin) {
  __shared__ unsigned int bin[1024];
  
  int tidx = threadIdx.x;

  bin[tidx] = d_histo[tidx];
  __syncthreads();

  // Reduce
  for (size_t w = 2; w <= 1024; w <<= 1) {
    if (tidx % w == w - 1)
      bin[tidx] += bin[tidx - w / 2];
    __syncthreads();
  }

  // Downsweep
  if (tidx == 1023)
    bin[1023] = 0;
  __syncthreads();

  for (size_t w = 1024; w >= 2; w >>= 1) {
    if (tidx % w == w - 1) {
      unsigned int s = bin[tidx - w / 2] + bin[tidx];
      bin[tidx - w / 2] = bin[tidx];
      bin[tidx] = s;
    }
    __syncthreads();
  }

  d_cdf[tidx] = bin[tidx];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  const size_t MAX_NUM_THREADS_PER_BLOCK = 1024;

  const size_t numPixels = numRows * numCols;

  int exponent = 0;
  while ((1 << (exponent + 1)) < numPixels)
    ++exponent;

  float *d_aux;
  checkCudaErrors(hipMalloc(&d_aux, sizeof(float) * numPixels));

  // Find min_logLum
  checkCudaErrors(hipMemcpy(d_aux,
                             d_logLuminance,
                             sizeof(float) * numPixels,
                             hipMemcpyDeviceToDevice));
  for (size_t width = 1 << exponent; width >= 1; width >>= 1) {
    const dim3 blockSize(MAX_NUM_THREADS_PER_BLOCK);
    const dim3 gridSize((width + MAX_NUM_THREADS_PER_BLOCK - 1) /
                        MAX_NUM_THREADS_PER_BLOCK);
    ReduceOnce<<<gridSize, blockSize>>>(d_aux, numPixels, width, 0);  // 0 - min
  }
  float h_min_logLum;
  checkCudaErrors(hipMemcpy(&h_min_logLum,
                             d_aux,
                             sizeof(float),
                             hipMemcpyDeviceToHost));
  min_logLum = h_min_logLum;

  // Find max_logLum
  checkCudaErrors(hipMemcpy(d_aux,
                             d_logLuminance,
                             sizeof(float) * numPixels,
                             hipMemcpyDeviceToDevice));
  for (size_t width = 1 << exponent; width >= 1; width >>= 1) {
    const dim3 blockSize(MAX_NUM_THREADS_PER_BLOCK);
    const dim3 gridSize((width + MAX_NUM_THREADS_PER_BLOCK - 1) /
                        MAX_NUM_THREADS_PER_BLOCK);
    ReduceOnce<<<gridSize, blockSize>>>(d_aux, numPixels, width, 1);  // 1 - max
  }
  float h_max_logLum;
  checkCudaErrors(hipMemcpy(&h_max_logLum,
                             d_aux,
                             sizeof(float),
                             hipMemcpyDeviceToHost));
  max_logLum = h_max_logLum;

  checkCudaErrors(hipFree(d_aux));

  // Find the range
  float logLumRange = max_logLum - min_logLum;

  // Build histogram
  unsigned int *d_histo;
  checkCudaErrors(hipMalloc(&d_histo, sizeof(unsigned int) * numBins));
  checkCudaErrors(hipMemset(d_histo, 0, sizeof(unsigned int) * numBins));
  {
    const dim3 blockSize(MAX_NUM_THREADS_PER_BLOCK);
    const dim3 gridSize((numPixels + MAX_NUM_THREADS_PER_BLOCK - 1) /
                        MAX_NUM_THREADS_PER_BLOCK);
    BuildHistogram<<<gridSize, blockSize>>>(d_logLuminance,
                                            numPixels,
                                            d_histo,
                                            numBins,
                                            min_logLum,
                                            logLumRange);
  }

  // Calculate the cumulative distribution
  {
    const dim3 blockSize(MAX_NUM_THREADS_PER_BLOCK);
    const dim3 gridSize(1);  // numBins = 1024 = MAX_NUM_THREADS_PER_BLOCK
    const size_t sharedMemoryBytes = sizeof(unsigned int) * numBins;
    ExclusivePrefixSum<<<gridSize, blockSize, sharedMemoryBytes>>>(d_histo,
                                                                   d_cdf,
                                                                   numBins);
  }

  checkCudaErrors(hipFree(d_histo));

  /****************************************************************************
  * You can use the code below to help with debugging, but make sure to       *
  * comment it out again before submitting your assignment for grading,       *
  * otherwise this code will take too much time and make it seem like your    *
  * GPU implementation isn't fast enough.                                     *
  *                                                                           *
  * This code generates a reference cdf on the host by running the            *
  * reference calculation we have given you.  It then copies your GPU         *
  * generated cdf back to the host and calls a function that compares the     *
  * the two and will output the first location they differ.                   *
  * ************************************************************************* */

  /*
  float *h_logLuminance = new float[numRows * numCols];
  unsigned int *h_cdf   = new unsigned int[numBins];
  unsigned int *h_your_cdf = new unsigned int[numBins];
  checkCudaErrors(hipMemcpy(h_logLuminance, d_logLuminance, numCols * numRows * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_your_cdf, d_cdf, numBins * sizeof(unsigned int), hipMemcpyDeviceToHost));

  referenceCalculation(h_logLuminance, h_cdf, numRows, numCols, numBins);

  //compare the results of the CDF
  checkResultsExact(h_cdf, h_your_cdf, numBins);
 
  delete[] h_logLuminance;
  delete[] h_cdf; 
  delete[] h_your_cdf;
  */
}
